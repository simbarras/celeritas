#include "hip/hip_runtime.h"
//----------------------------------*-C++-*----------------------------------//
// Copyright 2020-2023 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file celeritas/field/DormandPrinceStepper.cu
//---------------------------------------------------------------------------//
#include "DormandPrinceStepper.test.hh"

#include <typeinfo>

#include "corecel/sys/Device.hh"
#include "corecel/sys/KernelParamCalculator.device.hh"
#include "corecel/sys/ScopedProfiling.hh"
#include "celeritas/field/detail/FieldUtils.hh"

namespace celeritas
{
namespace test
{
//---------------------------------------------------------------------------//

namespace
{
//---------------------------------------------------------------------------//
// KERNELS
//---------------------------------------------------------------------------//
__global__ void test_kernel()
{
    // printf("Hello from block %d and thread %d\n", blockIdx.x, threadIdx.x);
    int i = threadIdx.x;
    int j = i;
    if (i < 4) {
        j = __shfl_down_sync(0x0000000f, i, 2);
        __syncthreads();
    }
    printf("Thread %d: before %d, after %d\n", threadIdx.x, i, j);
}

template<class Stepper_impl>
__device__ FieldStepperResult run_stepper(Stepper_impl stepper,
                                          int step,
                                          OdeState state,
                                          int id,
                                          int index,
                                          int num_states)
{
    return FieldStepperResult();
}

template<>
__device__ FieldStepperResult run_stepper(Stepper_uni stepper,
                                          int step,
                                          OdeState state,
                                          int id,
                                          int index,
                                          int num_states)
{
    if (index != 0) return FieldStepperResult();
    return stepper(step, state);
}

template<>
__device__ FieldStepperResult run_stepper(Stepper_multi stepper,
                                          int step,
                                          OdeState state,
                                          int id,
                                          int index,
                                          int num_states)
{
    // printf("thread %d, index %d\n", id, index);

    return stepper(step, state, id, index, num_states);
}

template<class Stepper_impl>
__global__ void dormand_test_arg_kernel(OdeState* states,
                                        FieldStepperResult* results,
                                        int* num_states,
                                        int* number_iterations,
                                        int* number_threads)
{
    constexpr double initial_step_size = 10000.0;
    constexpr double delta_chord = 1e-4;
    constexpr double half = 0.5;

    auto id = (blockIdx.x * blockDim.x + threadIdx.x) / *number_threads;

    if (id >= *num_states) return;

    auto index = (blockIdx.x * blockDim.x + threadIdx.x) % *number_threads;
    auto eval = make_dummy_equation(dormand_prince_dummy_field);
    Stepper_impl stepper{eval};
    FieldStepperResult res;
    auto state = states[id];
    auto step = initial_step_size;

    for (int i = 0; i < *number_iterations; ++i)
    {
        res = run_stepper(stepper,
                          step,
                          state,
                          id,
                          index,
                          *num_states);
        auto dchord
            = detail::distance_chord(state, res.mid_state, res.end_state);
        step *= max(std::sqrt(delta_chord / dchord), half);
    }
    results[id] = res;
}
} // namespace

//---------------------------------------------------------------------------//
// TESTING INTERFACE
//---------------------------------------------------------------------------//
//! Run on device and return results
void test()
{
    test_kernel<<<1, 32>>>();
}

KernelResult simulate_multi_next_chord(int number_threads)
{
    KernelResult result;

    // Load initial states and results to device
    int *d_num_states, *d_number_iterations, *d_number_threads;

    FieldStepperResult *h_results, *d_results;
    h_results = new FieldStepperResult[number_of_states];
    for (int i = 0; i < number_of_states; ++i)
    {
        h_results[i] = FieldStepperResult();
    }

    OdeState *d_states;

    // Create events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Allocate memory on device
    hipMalloc(&d_results, number_of_states * sizeof(FieldStepperResult));
    hipMalloc(&d_states, number_of_states * sizeof(OdeState));
    hipMalloc(&d_num_states, sizeof(int));
    hipMalloc(&d_number_iterations, sizeof(int));
    hipMalloc(&d_number_threads, sizeof(int));

    // Copy initial states to device
    hipMemcpy(d_states,
               initial_states,
               number_of_states * sizeof(OdeState),
               hipMemcpyHostToDevice);
    hipMemcpy(
        d_num_states, &number_of_states, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_number_iterations,
               &number_iterations,
               sizeof(int),
               hipMemcpyHostToDevice);
    hipMemcpy(
        d_number_threads, &number_threads, sizeof(int), hipMemcpyHostToDevice);

    // Kernel configuration
    int thread_dimension = number_threads * number_of_states;
    int shared_memory = number_of_states * 7 * sizeof(OdeState) +
                    number_of_states * sizeof(OdeState) +
                    number_of_states * sizeof(FieldStepperResult);

    // Launch the kernel
    hipEventRecord(start);
    if (number_threads > 1){
        dormand_test_arg_kernel<Stepper_multi>
            <<<1, thread_dimension, shared_memory>>>(d_states,
                                                       d_results,
                                                       d_num_states,
                                                       d_number_iterations,
                                                       d_number_threads);
    } else {
        dormand_test_arg_kernel<Stepper_uni>
            <<<1, thread_dimension>>>(d_states,
                                      d_results,
                                      d_num_states,
                                      d_number_iterations,
                                      d_number_threads);
    }

    hipDeviceSynchronize();
    hipEventRecord(stop);

    // Compute the elapsed time
    hipDeviceSynchronize();
    hipEventElapsedTime(&(result.milliseconds), start, stop);

    // Destroy events
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    // Copy results back to host
    hipMemcpy(h_results,
               d_results,
               number_of_states * sizeof(FieldStepperResult),
               hipMemcpyDeviceToHost);

    // Free memory on device
    hipFree(d_results);
    hipFree(d_states);
    hipFree(d_num_states);
    hipFree(d_number_iterations);
    hipFree(d_number_threads);

    // Return results
    result.results = h_results;
    return result;
}
//---------------------------------------------------------------------------//
}  // namespace test
}  // namespace celeritas
